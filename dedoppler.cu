#include "hip/hip_runtime.h"
#include <algorithm>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <functional>
#include <iostream>
#include <math.h>
#include <numeric>
#include <vector>

#include "cuda_util.h"
#include "dedoppler.h"
#include "taylor.h"
#include "util.h"

#include "fastDD_cpu.h"
#include "detection_fns.h"

# define DO_TAYLOR 0            // =1 Taylor =0 FastDD
# define DO_FASTDD_GPU 1        // =1 fastDD GPU, =0 fastDD CPU
# define FASTDD_N0 8            // fastDD first stage number of time steps

/*
  Gather information about the top hits.

  The eventual goal is for every frequency freq, we want:

  top_path_sums[freq] to contain the largest path sum that starts at freq
  top_drift_blocks[freq] to contain the drift block of that path
  top_path_offsets[freq] to contain the path offset of that path

  path_sums[path_offset][freq] contains one path sum.
  (In row-major order.)
  So we are just taking the max along a column and carrying some
  metadata along as we find it. One thread per freq.

  The function ignores data corresponding to invalid paths. See
  comments in taylor.cu for details.
*/
__global__ void findTopPathSums(const float* path_sums, int num_timesteps, int num_freqs,
                                int drift_block, float* top_path_sums,
                                int* top_drift_blocks, int* top_path_offsets) {
  int freq = blockIdx.x * blockDim.x + threadIdx.x;
  if (freq < 0 || freq >= num_freqs) {
    return;
  }

  for (int path_offset = 0; path_offset < num_timesteps; ++path_offset) {
    // Check if the last frequency in this path is out of bounds
    int last_freq = (num_timesteps - 1) * drift_block + path_offset + freq;
    if (last_freq < 0 || last_freq >= num_freqs) {
      // No more of these paths can be valid, either
      return;
    }

    float path_sum = path_sums[num_freqs * path_offset + freq];
    if (path_sum > top_path_sums[freq]) {
      top_path_sums[freq] = path_sum;
      top_drift_blocks[freq] = drift_block;
      top_path_offsets[freq] = path_offset;
    }
  }
}

/*
  Sum the columns of a two-dimensional array.
  input is a (num_timesteps x num_freqs) array, stored in row-major order.
  sums is an array of size num_freqs.
 */
__global__ void sumColumns(const float* input, float* sums, int num_timesteps, int num_freqs) {
  int freq = blockIdx.x * blockDim.x + threadIdx.x;
  if (freq < 0 || freq >= num_freqs) {
    return;
  }
  sums[freq] = 0.0;
  for (int i = freq; i < num_timesteps * num_freqs; i += num_freqs) {
    sums[freq] += input[i];
  }
}


/*
  The Dedopplerer encapsulates the logic of dedoppler search. In particular it manages
  the needed GPU memory so that we can reuse the same memory allocation for different searches.
 */
Dedopplerer::Dedopplerer(int num_timesteps, int num_channels, double foff, double tsamp,
                         bool has_dc_spike)
    : num_timesteps(num_timesteps), num_channels(num_channels), foff(foff), tsamp(tsamp),
      has_dc_spike(has_dc_spike), print_hits(false) {
  assert(num_timesteps > 1);
  rounded_num_timesteps = roundUpToPowerOfTwo(num_timesteps);

  #if DO_TAYLOR
    drift_timesteps = rounded_num_timesteps - 1;
        
    // Allocate everything we need for GPU processing 
    hipMalloc(&buffer1, num_channels * rounded_num_timesteps * sizeof(float));
    checkCuda("Dedopplerer buffer1 malloc");

    hipMalloc(&buffer2, num_channels * rounded_num_timesteps * sizeof(float));
    checkCuda("Dedopplerer buffer2 malloc");

  #else  // fastDD
    // leave buffer1 buffer2 allocation to within search
  
    drift_timesteps = rounded_num_timesteps;
  #endif

  drift_rate_resolution = 1e6 * foff / (drift_timesteps * tsamp);

  hipMalloc(&gpu_column_sums, num_channels * sizeof(float));
  hipHostMalloc(&cpu_column_sums, num_channels * sizeof(float));
  checkCuda("Dedopplerer column_sums malloc");
  
  hipMalloc(&gpu_top_path_sums, num_channels * sizeof(float));
  hipHostMalloc(&cpu_top_path_sums, num_channels * sizeof(float));
  checkCuda("Dedopplerer top_path_sums malloc");
   
  hipMalloc(&gpu_top_drift_blocks, num_channels * sizeof(int));
  hipHostMalloc(&cpu_top_drift_blocks, num_channels * sizeof(int));
  checkCuda("Dedopplerer top_drift_blocks malloc");
  
  hipMalloc(&gpu_top_path_offsets, num_channels * sizeof(int));
  hipHostMalloc(&cpu_top_path_offsets, num_channels * sizeof(int));
  checkCuda("Dedopplerer top_path_offsets malloc");
}

Dedopplerer::~Dedopplerer() {
  #if DO_TAYLOR
    hipFree(buffer1);
    hipFree(buffer2);
  #endif
  hipFree(gpu_column_sums);
  hipHostFree(cpu_column_sums);
  hipFree(gpu_top_path_sums);
  hipHostFree(cpu_top_path_sums);
  hipFree(gpu_top_drift_blocks);
  hipHostFree(cpu_top_drift_blocks);
  hipFree(gpu_top_path_offsets);
  hipHostFree(cpu_top_path_offsets);
}

// This implementation is an ugly hack
size_t Dedopplerer::memoryUsage() const {
  return num_channels * rounded_num_timesteps * sizeof(float) * 2
    + num_channels * (2 * sizeof(float) + 2 * sizeof(int));
}

/*
  Takes a bunch of hits that we found for coherent beams, and adds information
  about their incoherent beam

  Input should be the incoherent sum.
  This function re-sorts hits by drift, so be aware that it will change order.
 */
void Dedopplerer::addIncoherentPower(const FilterbankBuffer& input,
                                     vector<DedopplerHit>& hits) {
  assert(input.num_timesteps == rounded_num_timesteps);
  assert(input.num_channels == num_channels);

  sort(hits.begin(), hits.end(), &driftStepsLessThan);
  
  int drift_shift = rounded_num_timesteps - 1;
  
  // The drift block we are currently analyzing
  int current_drift_block = INT_MIN;

  // A pointer for the currently-analyzed drift block
  const float* taylor_sums = nullptr;

  // Note: fastDD has not been implemented here

  for (DedopplerHit& hit : hits) {
    // Figure out what drift block this hit belongs to
    int drift_block = (int) floor((float) hit.drift_steps / drift_shift);
    int path_offset = hit.drift_steps - drift_block * drift_shift;
    assert(0 <= path_offset && path_offset < drift_shift);

    // We should not go backwards
    assert(drift_block >= current_drift_block);

    if (drift_block > current_drift_block) {
      // We need to analyze a new drift block
      taylor_sums = optimizedTaylorTree(input.d_sg_data, buffer1, buffer2,
                                        rounded_num_timesteps, num_channels,
                                        drift_block);
      current_drift_block = drift_block;
    }

    long power_index = index2d(path_offset, hit.index, num_channels);
    assert(taylor_sums != nullptr);
    hipMemcpy(&hit.incoherent_power, taylor_sums + power_index,
               sizeof(float), hipMemcpyDeviceToHost);
  }
}

/*
  Runs dedoppler search on the input buffer.
  Output is appended to the output vector.
  
  All processing of the input buffer happens on the GPU, so it doesn't need to
  start off with host and device synchronized when search is called, it can still
  have GPU processing pending.
*/
void Dedopplerer::search(const FilterbankBuffer& input,
                         const FilterbankMetadata& metadata,
                         int beam, int coarse_channel,
                         double max_drift, double min_drift, double snr_threshold,
                         vector<DedopplerHit>* output) {
  assert(input.num_timesteps == rounded_num_timesteps);  // forces power of two
  assert(input.num_channels == num_channels);

  // Normalize the max drift in units of "horizontal steps per vertical step"
  double diagonal_drift_rate = drift_rate_resolution * drift_timesteps;
  double normalized_max_drift = max_drift / abs(diagonal_drift_rate);
  int min_drift_block = floor(-normalized_max_drift);
  int max_drift_block = floor(normalized_max_drift);

  int n_sti,n_lti,n_avg;
  float fs = metadata.foff*1e6; // FFT filter bank output sample rate prior to sti sum = bin bandwidth

  n_sti= MAX(1,abs(round(metadata.tsamp*fs)));
  n_lti = num_timesteps;
  n_avg = n_sti*n_lti;
  float xf = 1./n_avg/2.;

  int mid = num_channels / 2;

  printf("\ncoarse channel %d, n_sti=%d, n_lti=%d, n_avg=%d, Drift Blocks %d to %d\n",
          coarse_channel,n_sti,n_lti,n_avg,min_drift_block,max_drift_block);

  long start_ms = timeInMS();
  long start_ms_all = timeInMS();
  
  #if DO_TAYLOR  

    if (!input.managed) {
      // do explicit cpu to gpu copy for unmanaged sg buffers
      hipMemcpy(input.d_sg_data,input.sg_data,input.bytes,hipMemcpyHostToDevice);
      checkCuda("hipMemcpy-d_sg");
    }

    // This will create one cuda thread per frequency bin
    int grid_size = (num_channels + CUDA_MAX_THREADS - 1) / CUDA_MAX_THREADS;

    // Zero out the path sums in between each coarse channel because
    // we pick the top hits separately for each coarse channel
    hipMemsetAsync(gpu_top_path_sums, 0, num_channels * sizeof(float));

    sumColumns<<<grid_size, CUDA_MAX_THREADS>>>(input.d_sg_data, gpu_column_sums,
                                                rounded_num_timesteps, num_channels);
    checkCuda("sumColumns");

    double t_sumcols_sec = (timeInMS() - start_ms)*.001;
    start_ms = timeInMS();

    // Do the Taylor tree algorithm for each drift block
    for (int drift_block = min_drift_block; drift_block <= max_drift_block; ++drift_block) {
      // Calculate Taylor sums
      const float* taylor_sums = optimizedTaylorTree(input.d_sg_data, buffer1, buffer2,
                                                    rounded_num_timesteps, num_channels,
                                                    drift_block);

      // Track the best sums
      findTopPathSums<<<grid_size, CUDA_MAX_THREADS>>>(taylor_sums, rounded_num_timesteps,
                                                      num_channels, drift_block,
                                                      gpu_top_path_sums,
                                                      gpu_top_drift_blocks,
                                                      gpu_top_path_offsets);
      checkCuda("findTopPathSums");
    }

    // Now that we have done all the GPU processing for one coarse
    // channel, we can copy the data back to host memory.
    // These copies are not async, so they will synchronize to the default stream.
    hipMemcpy(cpu_column_sums, gpu_column_sums,
              num_channels * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(cpu_top_path_sums, gpu_top_path_sums,
              num_channels * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(cpu_top_drift_blocks, gpu_top_drift_blocks,
              num_channels * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(cpu_top_path_offsets, gpu_top_path_offsets,
              num_channels * sizeof(int), hipMemcpyDeviceToHost);
    checkCuda("dedoppler d->h memcpy");
    
    double t_DD_sec = (timeInMS() - start_ms)*.001;

  #else  // fastDD GPU or CPU

    /* Setup common to cpu and gpu versions */

    start_ms = timeInMS();

    DD_metadata *DD_meta,DD_meta1;
    DD_meta = &DD_meta1;

    double f_sg_min= metadata.fch1*1e6;  // SG start freq (bin center) Hz
    double df_sg = metadata.foff*1e6;    // SG freq increment Hz
    int Nf_sg = num_channels;            // SG number of frequency values
    double dt_sg = tsamp;                // SG time increment per line (time resolution, lines might be averaged)
    int Nt_sg = num_timesteps;           // SG number of lines (time values)
    double Lf = 1.0;            // PFB overlap factor (=1 for FFT filter bank) 
    int Nt = num_timesteps;    // total number of time samples to integrate (Nt/N0 is power of 2)   
    int N0 = MIN(FASTDD_N0,num_timesteps);  // first stage number of time samples   
    
    double dfdt_min_nom,dfdt_max_nom;
    uint max_DD_buffer_bytes = 0;
    
    /* Check all metadata for all drift blocks to be sure to allocate sufficient buffer sizes */

    for (int drift_block = min_drift_block; drift_block <= max_drift_block; ++drift_block) {
      dfdt_min_nom =  drift_block*df_sg/dt_sg; // desired DD minimum frequency rate Hz/sec (input)
      dfdt_max_nom =  (drift_block+1)*df_sg/dt_sg; // desired DD maximum frequency rate Hz/sec (input)
      gen_fastDD_metadata(DD_meta,f_sg_min,df_sg,Nf_sg,dt_sg,Nt_sg,dfdt_min_nom,dfdt_max_nom,Lf,Nt,N0);
      max_DD_buffer_bytes = MAX(DD_meta->DD_buffer_bytes,max_DD_buffer_bytes);
      //print_fastDD_metadata(DD_meta);
    }
    drift_rate_resolution = DD_meta->d_dfdt;

    printf("\n*** fastDD Buffer 1 2 bytes requirement: %u\n",max_DD_buffer_bytes);
    
    #if DO_FASTDD_GPU 

      float *gpu_det_DD;
      float *det_DD_work[2];

      // Allocatate buffers if first pass - sizing computed within gen_fastDD_metadata()
      // buffer1 & buffer2 for fastDD are slightly larger than SGs by a small factor
      
      hipMalloc(&buffer1, max_DD_buffer_bytes);
      checkCuda("Dedopplerer fastDD buffer1 malloc");
      hipMalloc(&buffer2, max_DD_buffer_bytes);
      checkCuda("Dedopplerer fastDD buffer2 malloc");
    
      if (!input.managed) {
        // do explicit cpu to gpu copy for unmanaged sg buffers
        hipMemcpy(input.d_sg_data,input.sg_data,input.bytes,hipMemcpyHostToDevice);
        checkCuda("hipMemcpy-d_sg");
      }

      // This will create one cuda thread per frequency bin
      int grid_size = (num_channels + CUDA_MAX_THREADS - 1) / CUDA_MAX_THREADS;
  
      det_DD_work[0] = buffer1;
      det_DD_work[1] = buffer2;
      
      // Zero out the path sums in between each coarse channel because
      // we pick the top hits separately for each coarse channel
      hipMemsetAsync(gpu_top_path_sums, 0, num_channels * sizeof(float));

      sumColumns<<<grid_size, CUDA_MAX_THREADS>>>(input.d_sg_data, gpu_column_sums,
                                                  rounded_num_timesteps, num_channels);
      
      checkCuda("sumColumns");
      double t_sumcols_sec = (timeInMS() - start_ms)*.001;
      printf("Sum Columns Elapsed time: %.2f sec\n",t_sumcols_sec);
      start_ms = timeInMS();

      for (int drift_block = min_drift_block; drift_block <= max_drift_block; ++drift_block) {

        dfdt_min_nom =  drift_block*df_sg/dt_sg; // desired DD minimum frequency rate Hz/sec (input)
        dfdt_max_nom =  (drift_block+1)*df_sg/dt_sg; // desired DD maximum frequency rate Hz/sec (input)
      
        gen_fastDD_metadata(DD_meta,f_sg_min,df_sg,Nf_sg,dt_sg,Nt_sg,dfdt_min_nom,dfdt_max_nom,Lf,Nt,N0);

        gpu_det_DD = fastDD_gpu(input.d_sg_data,det_DD_work,DD_meta);
      
        // hipDeviceSynchronize();
        // checkCuda("fastDD_gpu_return");

        // Track the best sums
        findTopPathSums2<<<grid_size, CUDA_MAX_THREADS>>>(gpu_det_DD, DD_meta->Nr,
                                                        num_channels, drift_block,
                                                        gpu_top_path_sums,
                                                        gpu_top_drift_blocks,
                                                        gpu_top_path_offsets);
        checkCuda("findTopPathSums2");
      }

      //hipDeviceSynchronize();
      // Now that we have done all the GPU processing for one coarse
      // channel, we can copy the data back to host memory.
      // These copies are not async, so they will synchronize to the default stream.
      hipMemcpy(cpu_column_sums, gpu_column_sums,
                num_channels * sizeof(float), hipMemcpyDeviceToHost);
      hipMemcpy(cpu_top_path_sums, gpu_top_path_sums,
                num_channels * sizeof(float), hipMemcpyDeviceToHost);
      hipMemcpy(cpu_top_drift_blocks, gpu_top_drift_blocks,
                num_channels * sizeof(int), hipMemcpyDeviceToHost);
      hipMemcpy(cpu_top_path_offsets, gpu_top_path_offsets,
                num_channels * sizeof(int), hipMemcpyDeviceToHost);
      checkCuda("dedoppler d->h memcpy");
      //hipDeviceSynchronize();
      
      double t_DD_sec = (timeInMS() - start_ms)*.001;
      printf("fastDD GPU Elapsed time: %.2f sec\n",t_DD_sec);

      hipFree(buffer1);
      hipFree(buffer2);

    #else   // run fastDD_cpu
    
      float *det_DD;
      float *det_DD_work[2];

      det_DD_work[0] = (float *) malloc(max_DD_buffer_bytes);
      det_DD_work[1] = (float *) malloc(max_DD_buffer_bytes);
      
      zeroTopPathSums_cpu(num_channels,cpu_top_path_sums,
                            cpu_top_drift_blocks,cpu_top_path_offsets);
      
      printf("\nTwo det_DD_work arrays allocated, each %.0f MBytes\n",max_DD_buffer_bytes/1024./1024.);
    
      double t_init_sec = (timeInMS() - start_ms)*.001;
      printf("Init Elapsed time: %.2f sec\n",t_init_sec);
      start_ms = timeInMS();
      
      for (int drift_block = min_drift_block; drift_block <= max_drift_block; ++drift_block) {
      
        dfdt_min_nom =  drift_block*df_sg/dt_sg; // desired DD minimum frequency rate Hz/sec (input)
        dfdt_max_nom =  (drift_block+1)*df_sg/dt_sg; // desired DD maximum frequency rate Hz/sec (input)
        
        gen_fastDD_metadata(DD_meta,f_sg_min,df_sg,Nf_sg,dt_sg,Nt_sg,dfdt_min_nom,dfdt_max_nom,Lf,Nt,N0);
      
        //print_fastDD_metadata(DD_meta);   
        
        /* run fastDD De-Doppler algorithm */

        det_DD = fastDD_cpu(input.sg_data,det_DD_work,DD_meta);
        
        //printf("drift_block %d, fast_DD complete\n",drift_block);

        /* find De-Doppler peaks in current drift block */
        
        findTopPathSums_cpu(det_DD,DD_meta->Nr,num_channels,
                            drift_block, cpu_top_path_sums,
                            cpu_top_drift_blocks,cpu_top_path_offsets);
      
      }
      double t_DD_sec = (timeInMS() - start_ms)*.001;
      printf("fastDD CPU Elapsed time: %.2f sec\n",t_DD_sec);

      /* find column sums in spectrogram */
      start_ms = timeInMS();
      sumColumns_cpu(input.sg_data, cpu_column_sums, num_timesteps, num_channels);

      double t_sumcols_sec = (timeInMS() - start_ms)*.001;
      printf("Sum Columns Elapsed time: %.2f sec\n",t_sumcols_sec);

      free(det_DD_work[0]);
      free(det_DD_work[1]);
    #endif  // fastDD GPU or CPU
  #endif // TaylorDD or fastDD

  /*
  ** Run special test averaging increasing durations, verify non-coh gain
  */

  #if 0
    ncoh_avg_test(input.sg_data, num_channels, num_timesteps, n_sti, 1);
    ncoh_avg_test(input.sg_data, num_channels, num_timesteps, n_sti, 8);
    ncoh_avg_test(input.sg_data, num_channels, num_timesteps, n_sti, 32);
    ncoh_avg_test(input.sg_data, num_channels, num_timesteps, n_sti, 128);
  #endif

  /*
  ** Find detections
  */

  start_ms = timeInMS();

  #if 1
    // remove DC bins
    cpu_column_sums[mid-3]=0.;
    cpu_column_sums[mid-2]=0.;
    cpu_column_sums[mid-1]=0.;
    cpu_column_sums[mid]=0.;
    cpu_column_sums[mid+1]=0.;
    cpu_column_sums[mid+2]=0.;
    cpu_column_sums[mid+3]=0.;
  #endif

  // normalize by number of averages
  for (int freq=0; freq<num_channels; freq++){
    cpu_column_sums[freq] *= xf;
    cpu_top_path_sums[freq] *= xf;
  }

  int n_subband = N_SUBBAND;
  int Nf_subband = num_channels/n_subband;
  if (Nf_subband<NF_SUBBAND_MIN) {
    n_subband = MAX(1,num_channels/NF_SUBBAND_MIN);
    Nf_subband = num_channels/n_subband;
  }
  float subband_mean[N_SUBBAND_MAX];
  float subband_std[N_SUBBAND_MAX];
  float subband_limit[N_SUBBAND_MAX];
  float subband_det_threshold[N_SUBBAND_MAX];
  float subband_m_std_ratio[N_SUBBAND_MAX];
  float *work;
  work = (float *) malloc(Nf_subband*sizeof(float));
  printf("\nNf=%d,n_subband=%d, Nf_subband=%d:\n",num_channels,n_subband,Nf_subband);

  if (n_subband==1){
    calc_mean_std_dev(cpu_column_sums,num_channels,&subband_mean[0],&subband_std[0]);
  } else {
    float shear_constant = 2.3;
    multipass_subband_mean_std(cpu_column_sums,num_channels,n_subband,shear_constant,
                work,subband_mean,subband_std,subband_limit);
  }
  
  for (int i_band=0; i_band<n_subband; i_band++){
    subband_m_std_ratio[i_band] = subband_mean[i_band]/subband_std[i_band];
    subband_det_threshold[i_band] = subband_mean[i_band] + snr_threshold*subband_std[i_band];
    if (i_band==0) {
        printf("subband=%d mean=%.0f std=%.0f mean/std=%.2f vs %.2f, snr_threshold=%.2f, det_thr=%.0f\n",
        i_band,subband_mean[i_band],subband_std[i_band],subband_m_std_ratio[i_band],sqrt(2*n_avg),
        snr_threshold,subband_det_threshold[i_band]);
    }
  }
  // examine mean/std ratio mean and std over all subbands
  // should have low variation (std should be small)
  float m_std_ratio_mean,m_std_ratio_std;
  if (n_subband>1) { 
    calc_mean_std_dev(subband_m_std_ratio,n_subband,&m_std_ratio_mean,&m_std_ratio_std);
    printf("n_subband=%d m_std_ratio_mean=%.2f vs %.2f m_std_ratio_std=%.2f\n\n",
          n_subband,m_std_ratio_mean,sqrt(2*n_avg),m_std_ratio_std);
  }
  
  float m,std_dev;
  calc_mean_std_dev(cpu_column_sums, num_channels, &m, &std_dev);

  double t_stats_sec = (timeInMS() - start_ms)*.001;
  start_ms = timeInMS();
    
  // We consider two hits to be duplicates if the distance in their
  // frequency indexes is less than window_size. We only want to
  // output the largest representative of any set of duplicates.
  // window_size is chosen just large enough so that a single bright
  // pixel cannot cause multiple hits.
  // First we break up the data into a set of nonoverlapping
  // windows. Any candidate hit must be the largest within this
  // window.
  int window_size = 2 * ceil(normalized_max_drift * drift_timesteps);

  if (coarse_channel==0) {
    printf("foff=%f MHz t_samp=%f sec, n_sti=%d, n_lti=%d, n_avg=%d, n_fft=%d\n",
            metadata.foff*1e6,metadata.tsamp,n_sti,n_lti,n_avg,num_channels);
    printf("drift_rate_resolution=%.3f drift_timesteps=%d diagonal_drift_rate=%.3f\n",
            drift_rate_resolution,drift_timesteps,diagonal_drift_rate);
    printf("max_drift=%.2f normalized_max_drift=%.2f drift_timesteps=%d window_size=%d=>%.0f Hz\n",
            max_drift,normalized_max_drift,drift_timesteps,window_size,window_size*fs);
    printf("Overall Coarse Channel mean=%6.0f std_dev=%6.0f mean/std=%6.3f vs %6.3f\n\n",
            m,std_dev,m/std_dev,sqrt(2*n_avg));
    }

  for (int i = 0; i * window_size < num_channels; ++i) {
    int candidate_freq = -1;

    int i_band = MIN(n_subband-1,((i+0.5) * window_size)/Nf_subband);
    float path_sum_threshold = subband_det_threshold[i_band];
    float local_mean = subband_mean[i_band];
    std_dev = subband_std[i_band];
  
    float candidate_path_sum = path_sum_threshold;

    for (int j = 0; j < window_size; ++j) {
      int freq = i * window_size + j;
      if (freq >= num_channels) {
        break;
      }
      if (cpu_top_path_sums[freq] > candidate_path_sum) {
        // This is the new best candidate of the window
        candidate_freq = freq;
        candidate_path_sum = cpu_top_path_sums[freq];
      }
    }
    if (candidate_freq < 0) {
      continue;
    }

    // Check every frequency closer than window_size if we have a candidate
    int window_end = min(num_channels, candidate_freq + window_size);
    bool found_larger_path_sum = false;
    for (int freq = max(0, candidate_freq - window_size + 1); freq < window_end; ++freq) {
      if (cpu_top_path_sums[freq] > candidate_path_sum) {
        found_larger_path_sum = true;
        break;
      }
    }
    if (!found_larger_path_sum) {
      // The candidate frequency is the best within its window
      int drift_bins = cpu_top_drift_blocks[candidate_freq] * drift_timesteps +
        cpu_top_path_offsets[candidate_freq];
      double drift_rate = drift_bins * drift_rate_resolution;
      float snr = (candidate_path_sum - local_mean) / std_dev;

      if ((abs(drift_rate) >= min_drift) && (abs(drift_rate)) <= max_drift) {
        DedopplerHit hit(metadata, candidate_freq, drift_bins, drift_rate,
                         snr, beam, coarse_channel, num_timesteps, candidate_path_sum);
        if (print_hits) {
          cout << "hit: " << hit.toString() << endl;
        }
        output->push_back(hit);
      }
    }
  }

  free(work);
  
  double t_log_hits_sec = (timeInMS() - start_ms)*.001;
  double t_search_sec = (timeInMS() - start_ms_all)*.001;

  printf("Elapsed times: coarse chnl %d, UM %d, fft %d, sti %d, lti %d\n",
              coarse_channel,(int)input.managed,num_channels,n_sti,n_lti);
  printf("Sum Columns:     %.3f sec\n",t_sumcols_sec);
  printf("Taylor GPU:      %.3f sec\n",t_DD_sec);
  printf("Stats:           %.3f sec\n",t_stats_sec);
  printf("Log Hits:        %.3f sec\n",t_log_hits_sec);
  printf("DeDoppler total: %.3f sec\n",t_search_sec);

}
